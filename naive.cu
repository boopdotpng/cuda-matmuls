#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector> 
#include "utils.h"

/*
this kernel accesses B the same way as row. 
*/
__global__ void matmul_B_row_strided(const float *a, const float *b, float *c) {
  uint row = blockIdx.y * blockDim.y + threadIdx.y;
  uint col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= N || col >= N) return;
  float sum = 0.0f;
  for (uint i = 0; i < N; i++) sum += a[row*N+i] * b[col*N+i];
  c[row*N+col] = sum;
}

__global__ void matmul_B_col_contiguous(const float *a, const float *b ,float *c) {
  uint row = blockIdx.y * blockDim.y + threadIdx.y;
  uint col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= N || col >= N) return;
  float sum = 0.0f;
  for (uint i = 0; i < N; i++) sum += a[row*N+i] * b[i*N+col];
  c[row*N+col] = sum;
}

int main(int argc, const char *argv[]) {
  buffers bufs = allocs();
  hipEvent_t start, stop;
  float gflops, ms;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N+15) / 16, (N+15)/16);
  printf("launching with %d,%d block dim\n", numBlocks.x, numBlocks.y);
  if (argc > 1) {
    hipEventRecord(start);
    matmul_B_col_contiguous<<<numBlocks, threadsPerBlock>>>(bufs.A, bufs.B, bufs.C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    gflops = calc_gflops(ms); 
  } 
  else {
    hipEventRecord(start);
    matmul_B_row_strided<<<numBlocks, threadsPerBlock>>>(bufs.A, bufs.B_t, bufs.C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    gflops = calc_gflops(ms); 
  } 

  bool valid = cpu_val(bufs);
  if (valid) printf("%s: %.2f gflops \n", (argc>1) ? "matmul b col contiguous" : "matmul b row strided", gflops);
  else printf("wrong.\n"); 
  return 0;
}