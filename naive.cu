#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector> 
#include <chrono>
#include "utils.h"

__global__ void matmul(const float *a, const float *b, float *c) {
  uint row = blockIdx.y * blockDim.y + threadIdx.y;
  uint col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= N || col >= N) return;
  float sum = 0.0f;
  for (uint i = 0; i < N; i++) sum += a[row*N+i] * b[col+i*N];
  c[row*N+col] = sum;
}

int main() {
  buffers bufs = allocs();
  Timer t;
  Timer t2;

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N+15) / 16, (N+15)/16);
  printf("launching with %d,%d block dim\n", numBlocks.x, numBlocks.y);
  t.begin();
  matmul<<<numBlocks, threadsPerBlock>>>(bufs.A, bufs.B, bufs.C);
  hipDeviceSynchronize();
  double gflops = t.end();

  t2.begin();
  bool valid = cpu_val(bufs);
  t2.end();
  if (valid) {
    printf("naive: %.2f gflops \n", gflops);
    printf("validation took %fs\n", t2.elapsed.count());
  }
  else printf("wrong.\n"); 
  return 0;
}